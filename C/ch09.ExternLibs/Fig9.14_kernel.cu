#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void cuda_kernel(int *A, int N) {
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < N)
    A[tid] += 1;
}

extern "C" {
void call_cuda_kernel(int *A, int N, hipStream_t s) {
  cuda_kernel<<<N, 1, 0, s>>>(A, N);
}
}

