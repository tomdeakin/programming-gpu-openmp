#include <hip/hip_runtime.h>

__global__ void cuda_kernel(int *A) {
  A[threadIdx.x + blockIdx.x * blockDim.x] += 1;
}

extern "C" {
void call_cuda_kernel(int *A, int N, hipStream_t s) {
  cuda_kernel<<<N, 16, 0, s>>>(A);
}
}

