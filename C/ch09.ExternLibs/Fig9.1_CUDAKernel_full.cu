
#include <hip/hip_runtime.h>
#include <stdio.h>

void use(double *A, double *B, double *C, int N) {
  printf("A[1024]=%d\n", A[1024]);
  printf("B[1024]=%d\n", B[1024]);
  printf("C[1024]=%d\n", C[1024]);
}

// A CUDA kernel
__global__ void init(double *A, int N, double val) {
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N)
    A[i] = val;
}

int main(void) {
  int N = 1024 * 1024;
  double *A, *B, *C;

  hipMalloc(&A, sizeof(double)*N);
  hipMalloc(&B, sizeof(double)*N);
  hipMalloc(&C, sizeof(double)*N);

  // Create a CUDA stream
  hipStream_t s;
  hipStreamCreate(&s);

  // Launch three CUDA kernels in the stream
  int blocks = N / 64;
  int threads = 64;
  init<<<blocks, threads, 0, s>>>(A, N, 1.0);
  init<<<blocks, threads, 0, s>>>(B, N, 2.0);
  init<<<blocks, threads, 0, s>>>(C, N, 0.0);

  // Wait for work in stream to finish
  hipStreamSynchronize(s);

  use(A,B,C,N); // 

  // Destroy stream
  hipStreamDestroy(s);
  hipFree(A);
  hipFree(B);
  hipFree(C);

  return 0;
}
