
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <assert.h>
#include <stdio.h>
#include <omp.h>

__global__ void cuda_kernel(int *A) {
  A[threadIdx.x + blockIdx.x * blockDim.x] += 1;
}

int main(void) {

  int N = 100000;
  int *A = (int *)malloc(sizeof(int) * N);
  #pragma omp target enter data map(alloc: A[:N])
  
  #pragma omp target teams loop nowait depend(out: A)
  for (int i = 0; i < N; ++i)
    A[i] = i;
  
  omp_interop_t iobj = omp_interop_none;
  #pragma omp interop init(targetsync: iobj) nowait depend(inout: A)
  
  // Check we have a CUDA runtime
  int err;
  assert(omp_get_interop_int(iobj, omp_ipr_fr_id, &err) == omp_ifr_cuda);
  
  // Get CUDA stream
  hipStream_t s = (hipStream_t) omp_get_interop_ptr(iobj, omp_ipr_targetsync, NULL);
  
  // Asynchronously enqueue CUDA kernel on the stream
  #pragma omp target data use_device_ptr(A)
  cuda_kernel<<<N, 16, 0, s>>>(A);
  
  #pragma omp interop use(iobj) nowait depend(inout: A)
  
  #pragma omp target teams loop nowait depend(inout: A)
  for (int i = 0; i < N; ++i)
    A[i] += 1;
  
  #pragma omp interop use(iobj) nowait depend(inout: A)
  
  #pragma omp target data use_device_ptr(A)
  cuda_kernel<<<N, 16, 0, s>>>(A);
  
  #pragma omp interop destroy(iobj) nowait depend(inout: A)
  
  #pragma omp taskwait

  #pragma omp target exit data map(from: A[:N])

  // Check solution
  for (int i = 0; i < N; ++i)
    assert(A[i] == i + 3);

  printf("Success\n");

  free(A);
}

